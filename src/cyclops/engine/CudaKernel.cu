#include <stdio.h>
#include <iostream>
#include <chrono>

#include <hipcub/hipcub.hpp>
#include "CudaKernel.h"

using namespace hipcub;

template <class T>
CudaKernel<T>::CudaKernel(T* h_in, int num_items)
{

    // Allocate device arrays
    hipMalloc(&d_in,  sizeof(T) * num_items);
    hipMalloc(&d_out, sizeof(T) * num_items);

    // Copy input from host to device
    hipMemcpy(d_in, h_in, sizeof(T) * num_items, hipMemcpyHostToDevice);

//    std::cout << "Created \n";
}

template <class T>
CudaKernel<T>::~CudaKernel()
{
//    std::cout << "Destroyed \n";
}

template <class T>
void CudaKernel<T>::CubScanMalloc(int num_items)
{
    // Determine temporary device storage requirements
    DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, d_in, d_out, num_items);

    // Allocate temporary storage
    hipMalloc(&d_temp_storage, temp_storage_bytes);
}

template <class T>
void CudaKernel<T>::CubScan(int num_items)
{
    DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, d_in, d_out, num_items);
}

template class CudaKernel<float>;
template class CudaKernel<double>;

