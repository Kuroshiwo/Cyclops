#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <chrono>
#include <hip/hip_vector_types.h>

#include <hipcub/hipcub.hpp>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/tuple.h>
#include <thrust/sequence.h>
#include <thrust/iterator/permutation_iterator.h>
//#include <thrust/transform_reduce.h>
//#include <thrust/for_each.h>

#include "CudaKernel.h"

using namespace hipcub;

template <typename RealType>
__global__ void kernelUpdateXBeta(int offX,
				  int offK,
				  const int taskCount,
				  RealType delta,
				  const RealType* d_X,
				  const int* d_K,
				  RealType* d_XBeta,
				  RealType* d_ExpXBeta,
				  RealType* d_Numerator,
				  RealType* d_Numerator2)
{
	int task = blockIdx.x * blockDim.x + threadIdx.x;

//	if (formatType == INDICATOR || formatType == SPARSE) {
	    int k = d_K[offK + task];
//	} else { // DENSE, INTERCEPT
//	    int k = task;
//	}

//	if (formatType == SPARSE || formatType == DENSE) {
//	    RealType inc = delta * d_X[offX + task];
//	} else { // INDICATOR, INTERCEPT
	    RealType inc = delta;
//	}

	if (task < taskCount) {
	    RealType xb = d_XBeta[k] + inc;
	    d_XBeta[k] = xb;
	    d_ExpXBeta[k] = exp(xb);
	    d_Numerator[k] = 0;
	    d_Numerator2[k] = 0;
	}
}

template <typename RealType>
__global__ void kernelUpdateXBeta1(int offX,
				  int offK,
				  const int taskCount,
				  int index,
				  const RealType* d_X,
				  const int* d_K,
				  double2* d_GH,
				  RealType* d_XjY,
				  RealType* d_Bound,
				  RealType* d_Beta,
				  RealType* d_XBeta,
				  RealType* d_ExpXBeta,
				  RealType* d_Numerator,
				  RealType* d_Numerator2)
{
	// process delta, update beta and bound
	double2 GH = *d_GH;
	RealType g = GH.x - d_XjY[index];
	RealType h = GH.y;
	RealType beta = d_Beta[index];

	RealType delta = -g/h; // no prior

	RealType bound = d_Bound[index];
	if (delta < -bound) {
		delta = -bound;
	} else if (delta > bound) {
		delta = bound;
	}
	d_Beta[index] = delta + beta;

	auto intermediate = max(2*abs(delta), bound/2);
	intermediate = max(intermediate, 0.001);
	d_Bound[index] = intermediate;


	// update xb and exb
	int task = blockIdx.x * blockDim.x + threadIdx.x;

//	if (formatType == INDICATOR || formatType == SPARSE) {
	    int k = d_K[offK + task];
//	} else { // DENSE, INTERCEPT
//	    int k = task;
//	}

//	if (formatType == SPARSE || formatType == DENSE) {
//	    RealType inc = delta * d_X[offX + task];
//	} else { // INDICATOR, INTERCEPT
	    RealType inc = delta;
//	}

	if (task < taskCount) {
	    RealType xb = d_XBeta[k] + inc;
	    d_XBeta[k] = xb;
	    d_ExpXBeta[k] = exp(xb);
	    d_Numerator[k] = 0;
	    d_Numerator2[k] = 0;
	}
}

template <typename RealType>
__global__ void kernelComputeNumeratorForGradient(int offX,
                                                  int offK,
                                                  const int taskCount,
                                                  const RealType* d_X,
                                                  const int* d_K,
                                                  RealType* d_ExpXBeta,
                                                  RealType* d_Numerator,
                                                  RealType* d_Numerator2)
{
	int task = blockIdx.x * blockDim.x + threadIdx.x;

//	if (formatType == INDICATOR || formatType == SPARSE) {
	    int k = d_K[offK + task];
//	} else { // DENSE, INTERCEPT
//	    int k = task;
//	}

	if (task < taskCount) {
//	    if (formatType == SPARSE || formatType == DENSE) {
//	        d_Numerator[k] = d_X[offX + task] * d_ExpXBeta[k];
//	        d_Numerator2[k] = d_X[offX + task] * d_Numerator[k];
//	    } else { // INDICATOR, INTERCEPT
	        d_Numerator[k] = d_ExpXBeta[k];
//	    }
	}
}

template <typename RealType>
__global__ void kernelProcessDelta(RealType* d_DeltaVector,
                                   RealType* d_Bound,
                                   RealType* d_Beta,
                                   RealType* d_XjY,
                                   double2* d_GH,
                                   int index)
{
	double2 GH = *d_GH;
	RealType g = GH.x - d_XjY[index];
	RealType h = GH.y;
	RealType beta = d_Beta[index];
	RealType delta = -g/h;
	RealType bound = d_Bound[index];

	if (delta < -bound) {
		delta = -bound;
	} else if (delta > bound) {
		delta = bound;
	}
	d_DeltaVector[index] = delta;
	d_Beta[index] = delta + beta;

	auto intermediate = max(2*abs(delta), bound/2);
	intermediate = max(intermediate, 0.001);
	d_Bound[index] = intermediate;
}


struct TuplePlus
{
	template<typename L, typename R>
	__host__ __device__
	thrust::tuple<L, L> operator()(thrust::tuple<L, L> lhs, thrust::tuple<R, R> rhs)
	{
		return thrust::make_tuple(thrust::get<0>(lhs) + thrust::get<0>(rhs), thrust::get<1>(lhs) + thrust::get<1>(rhs));
	}
};

struct TuplePlus3
{
	template<typename L, typename R>
	__host__ __device__
	thrust::tuple<L, L, L> operator()(thrust::tuple<L, L, L> lhs, thrust::tuple<R, R, R> rhs)
	{
		return thrust::make_tuple(thrust::get<0>(lhs) + thrust::get<0>(rhs),
							thrust::get<1>(lhs) + thrust::get<1>(rhs),
							thrust::get<2>(lhs) + thrust::get<2>(rhs));
	}
};

struct Double2Plus
{
	__host__ __device__
	double2 operator()(double2& a, double2& b)
	{
		double2 out;
		out.x = a.x + b.x;
		out.y = a.y + b.y;
		return out;
	}
};


template <typename RealType>
CudaKernel<RealType>::CudaKernel()
{
	std::cout << "CUDA class Created \n";
}

template <typename RealType>
CudaKernel<RealType>::~CudaKernel()
{
	hipFree(d_temp_storage0); // accDenom
	hipFree(d_temp_storage); // accNumer
	hipFree(d_temp_storage_acc); // accNAndD
	hipFree(d_temp_storage_gh); // cGAH
//	hipFree(d_init);
	std::cout << "CUDA class Destroyed \n";
}

template <typename RealType>
void CudaKernel<RealType>::allocTempStorage(thrust::device_vector<RealType>& d_Denominator,
					    thrust::device_vector<RealType>& d_Numerator,
					    thrust::device_vector<RealType>& d_Numerator2,
					    thrust::device_vector<RealType>& d_AccDenom,
					    thrust::device_vector<RealType>& d_AccNumer,
					    thrust::device_vector<RealType>& d_AccNumer2,
					    thrust::device_vector<RealType>& d_NWeight,
//					    thrust::device_vector<RealType>& d_Gradient,
//					    thrust::device_vector<RealType>& d_Hessian,
					    double2* dGH,
					    size_t& N,
					    thrust::device_vector<int>& indicesN)
{
//	thrust::sequence(indicesN.begin(), indicesN.end());

	// for scan in accDenom
	DeviceScan::InclusiveSum(d_temp_storage0, temp_storage_bytes0, &d_Denominator[0], &d_AccDenom[0], N);
	hipMalloc(&d_temp_storage0, temp_storage_bytes0);

	// for scan in accNumer
	auto results = thrust::make_zip_iterator(thrust::make_tuple(d_AccNumer.begin(), d_AccNumer2.begin()));
	auto begin = thrust::make_zip_iterator(thrust::make_tuple(d_Numerator.begin(), d_Numerator2.begin()));
	DeviceScan::InclusiveScan(d_temp_storage, temp_storage_bytes, begin, results, TuplePlus(), N);
	hipMalloc(&d_temp_storage, temp_storage_bytes);

        // for scan in compDAndN
        auto results_acc = thrust::make_zip_iterator(thrust::make_tuple(d_AccDenom.begin(), d_AccNumer.begin(), d_AccNumer2.begin()));
        auto begin_acc = thrust::make_zip_iterator(thrust::make_tuple(d_Denominator.begin(), d_Numerator.begin(), d_Numerator2.begin()));

        DeviceScan::InclusiveScan(d_temp_storage_acc, temp_storage_bytes_acc, begin_acc, results_acc, TuplePlus3(), N);
        hipMalloc(&d_temp_storage_acc, temp_storage_bytes_acc);

	// for reduction in compGAndH
	auto begin_gh = thrust::make_zip_iterator(thrust::make_tuple(d_NWeight.begin(),
											  d_AccNumer.begin(),
											  d_AccDenom.begin(),
											  d_AccNumer2.begin()));

	TransformInputIterator<double2, functorCGH<RealType>, ZipVec4> itr(begin_gh, cGAH);

	d_init.x = d_init.y = 0.0;

	DeviceReduce::Reduce(d_temp_storage_gh, temp_storage_bytes_gh, itr, dGH, N, Double2Plus(), d_init);
	hipMalloc(&d_temp_storage_gh, temp_storage_bytes_gh);

}

template <typename RealType>
void CudaKernel<RealType>::updateXBeta(const thrust::device_vector<RealType>& d_X,
				       const thrust::device_vector<int>& d_K,
				       unsigned int offX,
				       unsigned int offK,
				       const unsigned int taskCount,
				       RealType delta,
				       thrust::device_vector<RealType>& d_XBeta,
				       thrust::device_vector<RealType>& d_ExpXBeta,
				       thrust::device_vector<RealType>& d_Numerator,
				       thrust::device_vector<RealType>& d_Numerator2,
				       int gridSize, int blockSize)
{
	kernelUpdateXBeta<<<gridSize, blockSize>>>(offX,
		    			                       offK,
		    			                       taskCount,
		    			                       delta,
		    			                       thrust::raw_pointer_cast(&d_X[0]),
		    			                       thrust::raw_pointer_cast(&d_K[0]),
		    			                       thrust::raw_pointer_cast(&d_XBeta[0]),
		    			                       thrust::raw_pointer_cast(&d_ExpXBeta[0]),
		    			                       thrust::raw_pointer_cast(&d_Numerator[0]),
		    			                       thrust::raw_pointer_cast(&d_Numerator2[0]));
	hipDeviceSynchronize(); // MAS Wait until kernel completes; may be important for timing
}

template <typename RealType>
void CudaKernel<RealType>::updateXBeta1(const thrust::device_vector<RealType>& d_X,
				       const thrust::device_vector<int>& d_K,
				       unsigned int offX,
				       unsigned int offK,
				       const unsigned int taskCount,
				       double2* d_GH,
				       thrust::device_vector<RealType>& d_XjY,
				       thrust::device_vector<RealType>& d_Bound,
				       thrust::device_vector<RealType>& d_Beta,
				       thrust::device_vector<RealType>& d_XBeta,
				       thrust::device_vector<RealType>& d_ExpXBeta,
				       thrust::device_vector<RealType>& d_Numerator,
				       thrust::device_vector<RealType>& d_Numerator2,
				       int index,
				       int gridSize, int blockSize)
{
	kernelUpdateXBeta1<<<gridSize, blockSize>>>(offX,
		    			                       offK,
		    			                       taskCount,
		    			                       index,
		    			                       thrust::raw_pointer_cast(&d_X[0]),
		    			                       thrust::raw_pointer_cast(&d_K[0]),
							       d_GH,
							       thrust::raw_pointer_cast(&d_XjY[0]),
							       thrust::raw_pointer_cast(&d_Bound[0]),
							       thrust::raw_pointer_cast(&d_Beta[0]),
		    			                       thrust::raw_pointer_cast(&d_XBeta[0]),
		    			                       thrust::raw_pointer_cast(&d_ExpXBeta[0]),
		    			                       thrust::raw_pointer_cast(&d_Numerator[0]),
		    			                       thrust::raw_pointer_cast(&d_Numerator2[0]));
	hipDeviceSynchronize();
}

template <typename RealType>
void CudaKernel<RealType>::computeNumeratorForGradient(const thrust::device_vector<RealType>& d_X,
		    				    				       const thrust::device_vector<int>& d_K,
		    				    				       unsigned int offX,
		    				    				       unsigned int offK,
		    				    				       const unsigned int taskCount,
		    				    				       thrust::device_vector<RealType>& d_ExpXBeta,
		    				    				       thrust::device_vector<RealType>& d_Numerator,
		    				    				       thrust::device_vector<RealType>& d_Numerator2,
		    				    				       int gridSize, int blockSize)
{
	kernelComputeNumeratorForGradient<<<gridSize, blockSize>>>(offX,
		    				    				               offK,
		    				    				               taskCount,
		    				    				               thrust::raw_pointer_cast(&d_X[0]),
		    				    				               thrust::raw_pointer_cast(&d_K[0]),
		    				    				               thrust::raw_pointer_cast(&d_ExpXBeta[0]),
		    				    				               thrust::raw_pointer_cast(&d_Numerator[0]),
		    				    				               thrust::raw_pointer_cast(&d_Numerator2[0]));
	hipDeviceSynchronize(); // MAS Wait until kernel completes; may be important for timing
}


template <typename RealType>
void CudaKernel<RealType>::processDelta(thrust::device_vector<RealType>& d_DeltaVector,
		    				    		thrust::device_vector<RealType>& d_Bound,
		    				    		thrust::device_vector<RealType>& d_Beta,
		    				    		thrust::device_vector<RealType>& d_XjY,
		    				    		double2* d_GH,
		    				    		thrust::device_vector<RealType>& d_PriorParams,
		    				    		std::vector<RealType>& priorTypes,
		    				    		int index,
		    				    		int gridSize, int blockSize)
{
//	std::cout << "processDelta kernel \n";
	kernelProcessDelta<<<1, 1>>>(thrust::raw_pointer_cast(&d_DeltaVector[0]),
	                                            thrust::raw_pointer_cast(&d_Bound[0]),
	                                            thrust::raw_pointer_cast(&d_Beta[0]),
	                                            thrust::raw_pointer_cast(&d_XjY[0]),
	                                            d_GH,
	                                            index);
	hipDeviceSynchronize();
	d_DeltaVector[index];	
}

template <typename RealType>
void CudaKernel<RealType>::computeGradientAndHessian(thrust::device_vector<RealType>& d_AccNumer,
						     thrust::device_vector<RealType>& d_AccNumer2,
						     thrust::device_vector<RealType>& d_AccDenom,
						     thrust::device_vector<RealType>& d_NWeight,
//						     thrust::device_vector<RealType>& d_Gradient,
//						     thrust::device_vector<RealType>& d_Hessian,
						     double2* dGH,
						     size_t& N
//						     ,const std::vector<int>& K,
//                                                     unsigned int offK,
//                                                     thrust::device_vector<int>& indicesN
						     )
{
//	int start = K[offK];
/*
	for (int i = K[offK]; i < N; i++) {
	    std::cout << indicesN[i] << '\n';
	}
*/
	// cub transfrom reduction
	auto begin_gh = thrust::make_zip_iterator(thrust::make_tuple(d_NWeight.begin(),
                                            	                 d_AccNumer.begin(),
                                            	                 d_AccDenom.begin(),
                                            	                 d_AccNumer2.begin()));
	// transform iterator
	TransformInputIterator<double2, functorCGH<RealType>, ZipVec4> itr(begin_gh, cGAH);

	// Launch kernel
	DeviceReduce::Reduce(d_temp_storage_gh, temp_storage_bytes_gh, itr, dGH, N, Double2Plus(), d_init);
	hipDeviceSynchronize(); // MAS Wait until kernel completes; may be important for timing
/*
	// thrust::transform_reduce
	GH = thrust::transform_reduce(
                    thrust::make_zip_iterator(thrust::make_tuple(d_NWeight.begin(), d_AccNumer.begin(), d_AccDenom.begin(), d_AccNumer2.begin())),
                    thrust::make_zip_iterator(thrust::make_tuple(d_NWeight.end(), d_AccNumer.end(), d_AccDenom.end(), d_AccNumer2.end())),
                    cGAH,
                    d_init,
                    Double2Plus());

	// start from the first non-zero entry

	// Determine temporary device storage requirements and allocate temporary storage
	DeviceReduce::Reduce(d_temp_storage_gh, temp_storage_bytes_gh,
	    thrust::make_permutation_iterator(itr, indicesN.begin() + start),
	    results_gh, N, TuplePlus(), init);
	hipMalloc(&d_temp_storage_gh, temp_storage_bytes_gh);

	// Launch kernel
	DeviceReduce::Reduce(d_temp_storage_gh, temp_storage_bytes_gh,
	    thrust::make_permutation_iterator(itr, indicesN.begin() + start),
	    results_gh, N, TuplePlus(), init);
*/
}

template <typename RealType>
void CudaKernel<RealType>::computeAccumulatedNumerator(thrust::device_vector<RealType>& d_Numerator,
                                                       thrust::device_vector<RealType>& d_Numerator2,
                                                       thrust::device_vector<RealType>& d_AccNumer,
                                                       thrust::device_vector<RealType>& d_AccNumer2,
                                                       size_t& N)
{
	auto results = thrust::make_zip_iterator(thrust::make_tuple(d_AccNumer.begin(), d_AccNumer2.begin()));
	auto begin = thrust::make_zip_iterator(thrust::make_tuple(d_Numerator.begin(), d_Numerator2.begin()));

	// Launch kernel
	DeviceScan::InclusiveScan(d_temp_storage, temp_storage_bytes, begin, results, TuplePlus(), N);
	hipDeviceSynchronize(); // MAS Wait until kernel completes; may be important for timing
}

template <typename RealType>
void CudaKernel<RealType>::computeAccumulatedNumerAndDenom(thrust::device_vector<RealType>& d_Denominator,
                                                           thrust::device_vector<RealType>& d_Numerator,
                                                           thrust::device_vector<RealType>& d_Numerator2,
                                                           thrust::device_vector<RealType>& d_AccDenom,
                                                           thrust::device_vector<RealType>& d_AccNumer,
                                                           thrust::device_vector<RealType>& d_AccNumer2,
                                                           size_t& N)
{
    	auto results_acc = thrust::make_zip_iterator(thrust::make_tuple(d_AccDenom.begin(), d_AccNumer.begin(), d_AccNumer2.begin()));
    	auto begin_acc = thrust::make_zip_iterator(thrust::make_tuple(d_Denominator.begin(), d_Numerator.begin(), d_Numerator2.begin()));

     	// Launch kernel
    	DeviceScan::InclusiveScan(d_temp_storage_acc, temp_storage_bytes_acc, begin_acc, results_acc, TuplePlus3(), N);
	hipDeviceSynchronize(); // MAS Wait until kernel completes; may be important for timing
}

template <typename RealType>
void CudaKernel<RealType>::empty4(thrust::device_vector<RealType>& d_AccNumer,
				  thrust::device_vector<RealType>& d_AccNumer2,
				  thrust::device_vector<RealType>& d_Buffer1,
				  thrust::device_vector<RealType>& d_Buffer2)
{
	d_Buffer1 = d_AccNumer;
	d_Buffer2 = d_AccNumer2;
}

template <typename RealType>
void CudaKernel<RealType>::empty2(thrust::device_vector<RealType>& d_AccDenom,
                                  thrust::device_vector<RealType>& d_Buffer3)
{
	d_Buffer3 = d_AccDenom;
}


template <typename RealType>
void CudaKernel<RealType>::CubScan(RealType* d_in, RealType* d_out, int num_items)
{
	// Launch kernel
	DeviceScan::InclusiveSum(d_temp_storage0, temp_storage_bytes0, d_in, d_out, num_items);
	hipDeviceSynchronize(); // MAS Wait until kernel completes; may be important for timing
}


/* currently not using
template <typename RealType>
__global__ void kernelComputeGradientAndHessian(RealType* d_BufferG,
                                                RealType* d_BufferH,
                                                const RealType* d_AccNumer,
                                                const RealType* d_AccNumer2,
                                                const RealType* d_AccDenom,
                                                const RealType* d_NWeight,
                                                int N)
{
        int task = blockIdx.x * blockDim.x + threadIdx.x;

        if (task < N) {
                RealType t = d_AccNumer[task] / d_AccDenom[task];
                RealType g = d_NWeight[task] * t;
                d_BufferG[task] = g;
//          if (IteratorType::isIndicator) {
                        d_BufferH[task] = g * (1.0 - t);
//          } else {
//              d_BufferH[task] = d_NWeight[task] * (d_AccNumer2[task] / d_AccDenom[task] - t * t);
//          }
        }
}

template <typename RealType>
void CudaKernel<RealType>::CubReduce(RealType* d_in, RealType* d_out, int num_items)
{

    	// Declare temporary storage
    	void *d_temp_storage0 = NULL;
    	size_t temp_storage_bytes0 = 0;

    	// Determine temporary device storage requirements and allocate temporary storage
    	DeviceReduce::Sum(d_temp_storage0, temp_storage_bytes0, d_in, d_out, num_items);
    	hipMalloc(&d_temp_storage0, temp_storage_bytes0); // MAS Why?

    	// Launch kernel
    	DeviceReduce::Sum(d_temp_storage0, temp_storage_bytes0, d_in, d_out, num_items);

    	hipFree(d_temp_storage0);
}
*/

template class CudaKernel<float>;
template class CudaKernel<double>;

