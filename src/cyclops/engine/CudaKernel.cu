#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <chrono>

#include <hipcub/hipcub.hpp>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>

#include "CudaKernel.h"

using namespace hipcub;
	
template <typename T>
__global__ void kernelUpdateXBeta(int offX, int offK, const int taskCount, T delta,
                T* d_X, int* K, T* d_XBeta, T* d_ExpXBeta)
//__global__ void kernelUpdateXBeta(T* d_X, T* d_XBeta, T* d_ExpXBeta, T delta, int N)
{
    int task = blockIdx.x * blockDim.x + threadIdx.x;

    //if (formatType == INDICATOR || formatType == SPARSE) {
	int k = K[offK + task];
    //} else { // DENSE, INTERCEPT
//	int k = task;
    //}

    //if (formatType == SPARSE || formatType == DENSE) {
//	T inc = delta * d_X[offX + task];
    //} else { // INDICATOR, INTERCEPT
	T inc = delta;
    //}

    if (task < taskCount) {
	T xb = d_XBeta[k] + inc;
        d_XBeta[k] = xb;
	d_ExpXBeta[k] = expf(xb);
    }
}

template <typename T>
__global__ void kernelComputeGradientAndHessian(T* d_Gradient, T* d_Hessian, T* d_AccNumer, T* d_AccNumer2, T* d_AccDenom, T* d_NWeight, int N)
{
    int task = blockIdx.x * blockDim.x + threadIdx.x;

    if (task < N) {
        T t = d_AccNumer[task] / d_AccDenom[task];
        T g = d_NWeight[task] * t;
        d_Gradient[task] = g;
        //if (IteratorType::isIndicator) {
            d_Hessian[task] = g * (1.0 - t);
        //} else {
//	    d_Hessian[task] = d_NWeight[task] * (d_AccNumer2[task] / d_AccDenom[task] - t * t);
        //}
    }
}

template <class T>
CudaKernel<T>::CudaKernel(thrust::device_vector<T>& X, thrust::device_vector<int>& K, T* h_XBeta, T* h_ExpXBeta, int num_items)
{
//    std::cout << "X size: " << sizeof(X) << " T size: " << sizeof(T) << '\n';
//    std::cout << "K size: " << sizeof(K) << " int size: " << sizeof(int) << '\n';
    
    // Allocate device arrays
    hipMalloc(&d_XBeta,  sizeof(T) * num_items);
    hipMalloc(&d_ExpXBeta,  sizeof(T) * num_items);
    hipMalloc(&d_AccDenom, sizeof(T) * num_items);

    // Copy input from host to device
    d_X = thrust::raw_pointer_cast(&X[0]);
    d_K = thrust::raw_pointer_cast(&K[0]);
    hipMemcpy(d_XBeta, h_XBeta, sizeof(T) * num_items, hipMemcpyHostToDevice);
    hipMemcpy(d_ExpXBeta, h_ExpXBeta, sizeof(T) * num_items, hipMemcpyHostToDevice);
//    std::cout << "CUDA class Created \n";
}

template <class T>
CudaKernel<T>::CudaKernel(T* h_Numer, T* h_Numer2, T* h_AccDenom, T* h_NWeight, int num_items)
{
    // Allocate device arrays
    hipMalloc(&d_Numer,  sizeof(T) * num_items);
    hipMalloc(&d_Numer2,  sizeof(T) * num_items);
    hipMalloc(&d_AccDenom, sizeof(T) * num_items);
    hipMalloc(&d_NWeight, sizeof(T) * num_items);

    hipMalloc(&d_AccNumer, sizeof(T) * num_items);
    hipMalloc(&d_AccNumer2, sizeof(T) * num_items);
    hipMalloc(&d_Gradient, sizeof(T) * num_items);
    hipMalloc(&d_Hessian, sizeof(T) * num_items);
    hipMalloc(&d_G, sizeof(T));
    hipMalloc(&d_H, sizeof(T));


    // Copy input from host to device
    hipMemcpy(d_Numer, h_Numer, sizeof(T) * num_items, hipMemcpyHostToDevice);
    hipMemcpy(d_Numer2, h_Numer2, sizeof(T) * num_items, hipMemcpyHostToDevice);
    hipMemcpy(d_AccDenom, h_AccDenom, sizeof(T) * num_items, hipMemcpyHostToDevice);
    hipMemcpy(d_NWeight, h_NWeight, sizeof(T) * num_items, hipMemcpyHostToDevice);

//    std::cout << "CUDA class Created \n";
}

template <class T>
CudaKernel<T>::~CudaKernel()
{
//    std::cout << "CUDA class Destroyed \n";
}

template <class T>
void CudaKernel<T>::updateXBeta(unsigned int offX, unsigned int offK, const unsigned int taskCount, T delta, int gridSize, int blockSize)
{
//    auto start1 = std::chrono::steady_clock::now();

    kernelUpdateXBeta<<<gridSize, blockSize>>>(offX, offK, taskCount, delta, d_X, d_K, d_XBeta, d_ExpXBeta);

//    auto end1 = std::chrono::steady_clock::now();
//    timerG1 += std::chrono::duration<double, std::milli>(end1 - start1).count();
}

template <class T>
void CudaKernel<T>::computeGradientAndHessian(size_t& N, int& gridSize, int& blockSize)
{
//    auto start1 = std::chrono::steady_clock::now();

    kernelComputeGradientAndHessian<<<gridSize, blockSize>>>(d_Gradient, d_Hessian, d_AccNumer, d_AccNumer2, d_AccDenom, d_NWeight, N);

    CudaKernel<T>::CubReduce(d_Gradient, d_G, N);
    CudaKernel<T>::CubReduce(d_Hessian, d_H, N);

//    auto end1 = std::chrono::steady_clock::now();
//    timerG1 += std::chrono::duration<double, std::milli>(end1 - start1).count();
}

template <class T>
void CudaKernel<T>::CubReduce(T* d_in, T* d_out, int num_items)
{
    // Allocate temporary storage
    void *d_temp_storage0 = NULL;
    size_t temp_storage_bytes0 = 0;

    // Determine temporary device storage requirements
    DeviceReduce::Sum(d_temp_storage0, temp_storage_bytes0, d_in, d_out, num_items);

    // Allocate temporary storage
    hipMalloc(&d_temp_storage0, temp_storage_bytes0);

    // Launch kernel
    DeviceReduce::Sum(d_temp_storage0, temp_storage_bytes0, d_in, d_out, num_items);
    hipFree(d_temp_storage0);
}

template <class T>
void CudaKernel<T>::CubScan(T* d_in, T* d_out, int num_items)
{
    // Allocate temporary storage
    void *d_temp_storage0 = NULL;
    size_t temp_storage_bytes0 = 0;

    // Determine temporary device storage requirements
    DeviceScan::InclusiveSum(d_temp_storage0, temp_storage_bytes0, d_in, d_out, num_items);

    // Allocate temporary storage
    hipMalloc(&d_temp_storage0, temp_storage_bytes0);

    // Launch kernel
    DeviceScan::InclusiveSum(d_temp_storage0, temp_storage_bytes0, d_in, d_out, num_items);
    hipFree(d_temp_storage0);
}

template <class T>
void CudaKernel<T>::computeAccDenomMalloc(int num_items)
{
    // Determine temporary device storage requirements
    DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, d_ExpXBeta, d_AccDenom, num_items);

    // Allocate temporary storage
    hipMalloc(&d_temp_storage, temp_storage_bytes);
}

template <class T>
void CudaKernel<T>::computeAccDenom(int num_items)
{
    DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, d_ExpXBeta, d_AccDenom, num_items);
}

template <class T>
void CudaKernel<T>::computeAccNumerMalloc(int num_items)
{
    // Determine temporary device storage requirements
    DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, d_Numer, d_AccNumer, num_items);

    // Allocate temporary storage
    hipMalloc(&d_temp_storage, temp_storage_bytes);
}

template <class T>
void CudaKernel<T>::computeAccNumer(int num_items)
{
    DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, d_Numer, d_AccNumer, num_items);
    DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, d_Numer2, d_AccNumer2, num_items);
}

template <class T>
void CudaKernel<T>::CubExpScanMalloc(int num_items)
{
    // Determine temporary device storage requirements
    DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, d_XBeta, d_AccDenom, num_items);

    // Allocate temporary storage
    hipMalloc(&d_temp_storage, temp_storage_bytes);
}

template <class T>
void CudaKernel<T>::CubExpScan(int num_items)
{
//    auto start = std::chrono::steady_clock::now();

    TransformInputIterator<T, CustomExp, T*> d_itr(d_XBeta, exp_op);
    DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, d_itr, d_AccDenom, num_items);
    
//    auto end = std::chrono::steady_clock::now();
//    timerG += std::chrono::duration<double, std::milli>(end - start).count();
//    std::cout << "GPU takes " << timerG << " ms" << '\n';
}

template class CudaKernel<float>;
template class CudaKernel<double>;

